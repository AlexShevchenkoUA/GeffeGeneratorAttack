#include "hip/hip_runtime.h"

#include "math_functions.h"
#include "device_atomic_functions.h"

#include <fstream>
#include <iostream>
#include <string>
#include <ctime>
#include <iomanip>


using namespace std;


//Params for statistical hypothesis testing
const int Blocks = 14;
__device__ __constant__  int C = 133;
__device__ __constant__  int N = 32 * Blocks;
__device__ __constant__  int BLOCKS = Blocks;


//L1 Buffer
__device__ __managed__ unsigned int l1_buffer[8] = { 0 };
__device__ __managed__ unsigned int l1_gamma_buffer[8][Blocks] = { 0 };
__device__ __managed__ unsigned int l1_buffer_counter = 0;


//L2 Buffer
__device__ __managed__ unsigned int l2_buffer[24] = { 0 };
__device__ __managed__ unsigned int l2_gamma_buffer[24][Blocks] = { 0 };
__device__ __managed__ unsigned int l2_buffer_counter = 0;


//L3 Buffer
__device__ __managed__ unsigned int l3_buffer[16] = { 0 };
__device__ __managed__ unsigned int l3_buffer_counter = 0;


//LFSR structure
__device__ __constant__ unsigned int l1_feedback = 0b00110010100000000000000000000000;
__device__ __constant__ unsigned int l2_feedback = 0b01001000000000000000000000000000;
__device__ __constant__ unsigned int l3_feedback = 0b11110101000000000000000000000000;


//Service functions
unsigned int* read_data(string data_file, int& n);
bool gamma_cheack(const unsigned int* gamma, int n, unsigned int l1, unsigned int l2, unsigned int l3);
void lfsr_roll_back(unsigned int& l1, unsigned int& l2, unsigned int& l3, int n);
unsigned int host_parity(unsigned int n);
string bin(unsigned int);


//Kernals
__global__ void l1_register_brute_force(const unsigned int* gamma, const unsigned int state_prefix);
__global__ void l2_register_brute_force(const unsigned int* gamma, const unsigned int state_prefix);
__global__ void l3_register_brute_force(const unsigned int* gamma, const unsigned int l1_index, const unsigned int l2_index, const unsigned int state_prefix);



int main(int argc, char* argv[]) {
	//Reading data from file
	string file(argv[1]);
	int n = 0;
	unsigned int* host_gamma = read_data(file, n);
	unsigned int* device_gamma;
	unsigned int MAX_ROUND = n / Blocks;


	bool target = false;

	for (int i = 0; i < MAX_ROUND; i++) {
		//Duplication gamma from host memory to device memory
		hipMalloc(&device_gamma, Blocks * sizeof(unsigned int));
		hipMemcpy(device_gamma, &host_gamma[i * Blocks], Blocks * sizeof(unsigned int), hipMemcpyHostToDevice);

		//Brute force
		//Kernals params
		unsigned int threadsPerBlock = 1 << 5;
		unsigned int blocksPerGrig = 1 << 23;

		float progress = 0;
		
		cout << "Attack round: " << i << endl;

		//L1 brute force
		cout << "L1 brute force: " << endl;
		for (unsigned int r = 0; r < 4; r++) {
			l1_register_brute_force <<< blocksPerGrig, threadsPerBlock >>> (device_gamma, r);
			hipDeviceSynchronize();
			progress = (((float)r) / 4) * 100;
			cout << fixed << setprecision(2) << "\r\t[Progress: " << progress << "%]";
		}
		cout << "\r\t[Progress: complete]" << endl;

		//L2 brute force
		cout << "L2 brute force: " << endl;
		for (unsigned int r = 0; r < 8; r++) {
			l2_register_brute_force <<< blocksPerGrig, threadsPerBlock >>> (device_gamma, r);
			hipDeviceSynchronize();
			progress = (((float)r) / 8) * 100;
			cout << fixed << setprecision(2) << "\r\t[Progress: " << progress << "%]";
		}
		cout << "\r\t[Progress: complete]" << endl;

		//L3 brute force
		unsigned int J, K;
		hipMemcpy(&J, &l1_buffer_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(&K, &l2_buffer_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);


		if ((J == 0) || (J >= 8)) {
			unsigned int zero = 0;
			hipMemcpy(&l1_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
			hipMemcpy(&l2_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
			cout << "L3 brute force: [failed]" << endl;
			continue;
		}
		if ((K == 0) || (K >= 24)) {
			unsigned int zero = 0;
			hipMemcpy(&l1_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
			hipMemcpy(&l2_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
			cout << "L3 brute force: [failed]" << endl;
			continue;
		}


		hipDeviceSynchronize();


		cout << "L3 brute force:" << endl;


		for (int j = 0; j < J; j++) {
			for (int k = 0; k < K; k++) {
				//L3 brute force kernal
				for (int r = 0; r < 16; r++) {
					l3_register_brute_force <<< blocksPerGrig, threadsPerBlock >>> (device_gamma, j, k, r);
					hipDeviceSynchronize();
					progress = (((float)(j * 16 * K + k * 16 + r)) / (J * K * 16)) * 100;
					cout << fixed << setprecision(2) << "\r\t[Progress: " << progress << "%]";
				}

				//Gamma cheack
				unsigned int M;
				hipMemcpy(&M, &l3_buffer_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);
				if ((M == 0) || (M >= 16)) {
					continue;
				}
				unsigned int l1_state, l2_state, l3_state;
				hipMemcpy(&l1_state, &l1_buffer[j], sizeof(unsigned int), hipMemcpyDeviceToHost);
				hipMemcpy(&l2_state, &l2_buffer[k], sizeof(unsigned int), hipMemcpyDeviceToHost);

				for (int m = 0; m < M; m++) {
					hipMemcpy(&l3_state, &l3_buffer, sizeof(unsigned int), hipMemcpyDeviceToHost);
					lfsr_roll_back(l1_state, l2_state, l3_state, i * 32 * Blocks);
					target = gamma_cheack(host_gamma, 32 * n, l1_state, l2_state, l3_state);
					if (target) {
						cout << "\r\t[Progress: complete]" << endl;
						cout << "Result:" << endl;
						cout << "L1 state: " << bin(l1_state) << " (" << hex << l1_state << ")" << endl;
						cout << "L2 state: " << bin(l2_state) << " (" << hex << l2_state << ")" << endl;
						cout << "L3 state: " << bin(l3_state) << " (" << hex << l3_state << ")" << endl;
						break;
					}
				}

				//Params reset
				M = 0;
				hipMemcpy(&l3_buffer_counter, &M, sizeof(unsigned int), hipMemcpyHostToDevice);

				if (target)
					break;
			}

			if (target)
				break;
		}
		if (target)
			break;

		cout << "\r\t[Progress: complete]" << endl;
		cout << "Attack round: [failed]" << endl;


		//Params reset
		unsigned int zero = 0;
		hipMemcpy(&l1_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(&l2_buffer_counter, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);

		//Memory free
		hipFree(device_gamma);
	}

	system("pause");

	//Memory free
	delete[] host_gamma;
	hipFree(device_gamma);

	return 0;
}


void lfsr_roll_back(unsigned int& l1, unsigned int& l2, unsigned int& l3, int n) {
	unsigned int l1_fb = 0b00110010100000000000000000000000;
	unsigned int l2_fb = 0b01001000000000000000000000000000;
	unsigned int l3_fb = 0b11110101000000000000000000000000;

	if (n == 0)
		return;

	for (int i = 0; i < n; i++) {
		unsigned int out_1, out_2, out_3;

		out_1 = l1 & 0x1;
		out_2 = l2 & 0x1;
		out_3 = l3 & 0x1;

		l1 = l1 >> 1;
		l2 = l2 >> 1;
		l3 = l3 >> 1;

		if (host_parity(l1 & l1_fb) != out_1)
			l1 ^= (1 << 29);
		if (host_parity(l2 & l2_fb) != out_2)
			l2 ^= (1 << 30);
		if (host_parity(l3 & l3_fb) != out_3)
			l3 ^= (1 << 31);
	}
}


string bin(unsigned int n) {
	string res = "";
	for (int i = 0; i < 32; i++) {
		if (i % 4 == 0)
			res = " " + res;
		if ((n & 0x1) == 1)
			res = '1' + res;
		else
			res = '0' + res;
		n = n >> 1;
	}
	return res;
}


inline __device__ unsigned int parity(unsigned int x) {
	return __popc(x) & 0b1;
}


inline __device__ unsigned int weigth(unsigned int x) {
	return __popc(x);
}


unsigned int* read_data(string data_file, int& n) {
	string data;

	ifstream in(data_file);
	in >> data;
	in.close();

	n = ceill(data.length() / 32);

	unsigned int* gamma = new unsigned int[n];

	for (int i = 0; i < n; i++)
		gamma[i] = 0;

	for (int i = 0; i < data.length(); i++)
		if (data[i] == '1')
			gamma[int(i / 32)] += (1 << (i % 32));

	return gamma;
}


bool gamma_cheack(const unsigned int* gamma, int n, unsigned int l1, unsigned int l2, unsigned int l3) {
	unsigned int l1_fb = 0b00110010100000000000000000000000;
	unsigned int l2_fb = 0b01001000000000000000000000000000;
	unsigned int l3_fb = 0b11110101000000000000000000000000;

	for (int i = 0; i < (int(n / 32)); i++) {
		unsigned int l1_gamma = 0;
		unsigned int l2_gamma = 0;
		unsigned int l3_gamma = 0;

		unsigned int out;

		for (int j = 0; j < 32; j++) {
			out = host_parity(l1 & l1_fb);
			l1_gamma += (((l1 >> 29) & 0x1) << j);
			l1 = (l1 << 1) ^ out;
			out = host_parity(l2 & l2_fb);
			l2_gamma += (((l2 >> 30) & 0x1) << j);
			l2 = (l2 << 1) ^ out;
			out = host_parity(l3 & l3_fb);
			l3_gamma += (((l3 >> 31) & 0x1) << j);
			l3 = (l3 << 1) ^ out;
		}

		unsigned int temp = (((l1_gamma ^ l2_gamma) & l3_gamma) ^ l2_gamma) ^ gamma[i];
		if (temp != 0)
			return false;
	}
	return true;
}


unsigned int host_parity(unsigned int n) {
	n = (n >> 16) ^ n;
	n = (n >> 8) ^ n;
	n = (n >> 4) ^ n;
	n = (n >> 2) ^ n;
	n = (n >> 1) ^ n;
	return (n & 0x1);
}


__global__ void l1_register_brute_force(const unsigned int* gamma, const unsigned int state_prefix) {
	unsigned int s = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
	unsigned int R = 0;

	if (s == 0)
		return;

	for (int i = 0; i < BLOCKS; i++) {
		unsigned int temp = 0;

		for (int j = 0; j < 32; j++) {
			unsigned int out = parity(s & l1_feedback);
			temp += (((s >> 29) & 0x1) << j);
			s = (s << 1) ^ out;
		}

		R += weigth(gamma[i] ^ temp);
	}

	if (R < C) {
		unsigned int i = atomicAdd(&l1_buffer_counter, 1);

		if (i >= 8)
			return;

		s = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
		l1_buffer[i] = s;

		for (int j = 0; j < BLOCKS; j++) {
			unsigned int temp = 0;

			for (int k = 0; k < 32; k++) {
				unsigned int out = parity(s & l1_feedback);
				temp += (((s >> 29) & 0x1) << k);
				s = (s << 1) ^ out;
			}

			l1_gamma_buffer[i][j] = temp;
		}
	}
}


__global__ void l2_register_brute_force(const unsigned int* gamma, const unsigned int state_prefix) {
	unsigned int s = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
	unsigned int R = 0;

	if (s == 0)
		return;

	for (int i = 0; i < BLOCKS; i++) {
		unsigned int temp = 0;

		for (int j = 0; j < 32; j++) {
			unsigned int out = parity(s & l2_feedback);
			temp += (((s >> 30) & 0x1) << j);
			s = (s << 1) ^ out;
		}

		R += weigth(gamma[i] ^ temp);
	}

	if (R < C) {
		unsigned int i = atomicAdd(&l2_buffer_counter, 1);
		if (i >= 24)
			return;

		s = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
		l2_buffer[i] = s;

		for (int j = 0; j < BLOCKS; j++) {
			unsigned int temp = 0;

			for (int k = 0; k < 32; k++) {
				unsigned int out = parity(s & l2_feedback);
				temp += (((s >> 30) & 0x1) << k);
				s = (s << 1) ^ out;
			}

			l2_gamma_buffer[i][j] = temp;
		}
	}
}


__global__ void l3_register_brute_force(const unsigned int* gamma, const unsigned int l1_index, const unsigned int l2_index, const unsigned int state_prefix) {
	unsigned int s = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
	if (s == 0)
		return;

	for (int i = 0; i < BLOCKS; i++) {
		unsigned int temp = 0;

		for (int j = 0; j < 32; j++) {
			unsigned int out = parity(s & l3_feedback);
			temp += (((s >> 31) & 0x1) << j);
			s = (s << 1) ^ out;
		}

		temp = (((l1_gamma_buffer[l1_index][i] ^ l2_gamma_buffer[l2_index][i]) & temp) ^ l2_gamma_buffer[l2_index][i]) ^ gamma[i];
		if (temp != 0)
			return;
	}

	unsigned int i = atomicAdd(&l3_buffer_counter, 1);
	if (i >= 16)
		return;

	l3_buffer[i] = (state_prefix << 28) ^ (blockIdx.x << 5) ^ (threadIdx.x);
}